/*****************************************************
 * This file tests cuda memory management APIs.
 *****************************************************/
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>


/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 *
 * See cuda.h for error code descriptions.
 */
#define CHECK_CUDA_RESULT(N) {											\
	hipError_t result = N;												\
	if (result != 0) {													\
		printf("CUDA call on line %d returned error %d\n", __LINE__,	\
			result);													\
		printf("Error: %s.\n", hipGetErrorString(result));             \
		exit(1);														\
	} }

/*
 * Initialzing and computing kernels
 */
__global__ void vecInit(float *A, float value) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("In GPU: %f.\n", value);
	A[i] = value;

}

__global__ void vecAdd(float* A, float* B, float* C) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	C[i] = A[i] + B[i];
	//printf("In GPU:\n");
}

__global__ void vecMultiply(float* A, float* B, float* C) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	C[i] = A[i] * B[i];
}

__global__ void vecMultiplyAndAdd(float* A, float* B, float* C, float* D) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	D[i] = (A[i] + B[i]) * C[i];
}

double currentTimeCPUSecond() {
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

void test_cudaDeviceGetAttribute(int* value, hipDeviceAttribute_t attr, int device) {
	hipDeviceGetAttribute(value, attr, device);
}

void initialData(float *A, unsigned int n, float data, int mode = 0) {
	//Using GPU to initialize the data, so that no need to copy memory from GPU to CPU
	//printf("Initialize the data.\n");
	if (mode == 0) {
		dim3 threadsPerBlock(1024);
		dim3 numBlocks((n+threadsPerBlock.x-1) / threadsPerBlock.x);
		//printf("Threads per block: %d, Blocks: %d.\n", threadsPerBlock.x, numBlocks.x);
		vecInit<<<numBlocks, threadsPerBlock>>>(A, data);
		hipDeviceSynchronize();
	} else {
		unsigned int i;
		for (i = 0; i < n; i++) {
			A[i] = data;
		}
	}
}

void profileAddtion (unsigned int nElement, unsigned int totalLoop) {
	unsigned int nBytes = nElement * sizeof(float);

	if (nBytes < 1024) {
		printf("Allocate nbytes is %d B.\n", nBytes);
	} else if (nBytes >= 1024 && nBytes < 1024*1024) {
		printf("Allocate nbytes is %d KB.\n", nBytes/1024);
	} else {
		printf("Allocate nbytes is %d MB.\n", nBytes/(1024*1024));
	}

	// allocate memory
	float *g_A[64*1024], *g_B[64*1024], *g_C[64*1024];

	for (int loop=0; loop<totalLoop; loop++) {
		printf("==== ==== ==== ==== Loop: %d.\n", loop);

		// unsigned int flags = cudaMemAttachHost;
		unsigned int flags = hipMemAttachGlobal;
		CHECK_CUDA_RESULT(hipMallocManaged(&g_A[loop], nBytes, flags));
		CHECK_CUDA_RESULT(hipMallocManaged(&g_B[loop], nBytes, flags));
		CHECK_CUDA_RESULT(hipMallocManaged(&g_C[loop], nBytes, flags));

		printf("===== inital data begins...\n");
		int mode = 0;
		double iStart = currentTimeCPUSecond();
		initialData(g_A[loop], nElement, 2.0f, mode);
		initialData(g_B[loop], nElement, 2.0f, mode);
		initialData(g_C[loop], nElement, 0.0f, mode);
		double iStop = currentTimeCPUSecond();
		if (mode == 0) {
			printf("==== GPU mode: time for initializing the data: %f.\n", iStop - iStart);
		} else {
			printf("==== CPU mode: time for initializing the data: %f.\n", iStop - iStart);
		}

		printf("===== add data begins...\n");
		iStart = currentTimeCPUSecond();
		dim3 threadsPerBlock(1024);
		dim3 numBlocks((nElement+threadsPerBlock.x-1) / threadsPerBlock.x);
		vecAdd<<<numBlocks, threadsPerBlock>>>(g_A[loop], g_B[loop], g_C[loop]);
		//cudaMemcpy(g_C[loop], g_A[loop], nElem, cudaMemcpyDeviceToDevice);
		hipDeviceSynchronize();
		iStop = currentTimeCPUSecond();
		printf("==== Time for adding the data: %f.\n", iStop - iStart);
	}

	//Check the accuracy
	float ans = 4.0f;
	//printf("===== ans is %f\n", ans);
	int ii, jj;
	for (ii = 0; ii < totalLoop; ii++) {
		for (jj = 0; jj < nElement; jj++) {
			if ((g_C[ii])[jj] != ans)
			{
				printf("Error happens, should enable DEBUG mode to investigate.\n");
				break;
			}
		}
	}

	if(ii==totalLoop && jj==nElement) {
		printf("Testing is passed.\n");
	}

#ifdef DEBUG
	printf("===== Check the results...\n");

	for (int i = 0; i < totalLoop; i++) {
		printf("\n======================================================\n");
		for (int j = 0; j < 8; j++) {
			//if ((g_C[i])[j] != ans)
			{
				printf("%3.0f ", (g_A[i])[j]);
			}
		}
	}
	printf("\n");

	float ans = 4.0f;
	printf("===== ans is %f\n", ans);
	for (int i = 0; i < totalLoop; i++) {
		printf("\n======================================================\n");
		for (int j = 0; j < 8; j++) {
			//if ((g_C[i])[j] != ans)
			{
				printf("%3.0f ", (g_C[i])[j]);
			}
		}
	}
	printf("\n");
#endif

	for (int i = 0; i < totalLoop; i++) {
		hipFree(g_A[i]);
		hipFree(g_B[i]);
		hipFree(g_C[i]);
	}
}



void test_cudaMallocManaged(int dev, int ipower) {
	int val;

	// Check if supports managed memory
	CHECK_CUDA_RESULT(hipDeviceGetAttribute(&val, hipDeviceAttributeManagedMemory, dev));

	// Check concurrent managed access, for cuda 8.0
	hipDeviceGetAttribute(&val, hipDeviceAttributeConcurrentManagedAccess, dev);
	if (!val) {
		printf("*** Warn: Concurrent managed access is not supported!\n");
	}

	int totalLoop = 1024*2;
	//int totalLoop = 10;

	profileAddtion(16*1024*1024, totalLoop);

	hipDeviceReset();
}

int main(int argc, char* argv[]) {

	// set up device
	int dev = 0;
	hipSetDevice(dev);

	// get device properties
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	// check uva supporting
	if (deviceProp.unifiedAddressing) {
		printf("Device %d supports uva memory!\n", dev);
	} else {
		printf("Device %d does not support uva memory!\n", dev);
		exit(EXIT_SUCCESS);
	}

	// set up date size of vectors
	int ipower = 20+4;
	if (argc > 1)
		ipower = atoi(argv[1]);

	test_cudaMallocManaged(dev, ipower);

}

